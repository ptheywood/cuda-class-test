#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDACHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

class TestClass {

public:
    int * data;
    size_t len;



    TestClass(size_t len) {
        printf("Constructor\n");
        this->data = nullptr;
        this->len = len;
    }

    ~TestClass(){
        printf("~Destructor\n");
    }

    __host__ void allocate(){
        CUDACHECK(hipMalloc((void**) &this->data, this->len * sizeof(int)));
        CUDACHECK(hipMemset(this->data, 0, this->len * sizeof(int)));
    }

    __host__ void free(){
        CUDACHECK(hipFree(this->data));
        this->data = nullptr;
    }

    __device__ int get(size_t index){
        return this->data[index];
    }
    __device__ void set(size_t index, int value){
        this->data[index] = value;
    }

};

__global__ void twodims_kernel(unsigned int maxx, unsigned int maxy){
    unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

    unsigned int gid = col + (row * (blockDim.x * gridDim.x));

    if(col < maxx && row < maxy){
        // Only print in some threads. 
        if (gid < 8){
            printf("gid %u, col: %u, row: %u valid\n", gid, col, row);
        }
        
    } else {
        if(gid < 8){
            printf("gid %u, col: %u, row: %u bad\n", gid, col, row);
        }
    }
}


void launch2dexample(){
    printf("launch2dexample\n");
    unsigned int XLEN = 8;
    unsigned int YLEN = 4;

    printf("problem size of %u x %u\n", XLEN, YLEN);

    unsigned int totalElements = XLEN * YLEN;
    
    // suggest block dimensions. Threads per block must not exceed 1024 on most hardware, registers will probably be a limiting factor. 
    dim3 blocksize(2, 2);

    // shrink either if larger than the actual dimensions to minimise work
    if(blocksize.x > XLEN){
        blocksize.x = XLEN;
    }
    if(blocksize.y > YLEN){
        blocksize.y = YLEN;
    }

    dim3 gridsize;
    gridsize.x = (XLEN + blocksize.x - 1) / blocksize.x;
    gridsize.y = (YLEN + blocksize.y - 1) / blocksize.y;

    unsigned int totalThreads = (blocksize.x * blocksize.y) * (gridsize.x * gridsize.y);


    printf("Launching %d x %d threads per block, with %d x %d blocks.\n %u elements, %u threads\n",
        blocksize.x, blocksize.y, gridsize.x, gridsize.y, totalElements, totalThreads);

    // Launch the kernel. 
    twodims_kernel<<<blocksize, gridsize, 0, 0>>>(XLEN, YLEN);

    // synchronize after the kernel to make sure there were no errors. 
    CUDACHECK(hipDeviceSynchronize());
    printf("launch2dexample finished\n");

}



__global__ void test_kernel(unsigned int threads, TestClass * d_instance){
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < threads){
        // printf("Thread %u\n", tid);
        printf("Thread %u: d_isntance %p, element %d\n", tid, d_instance, d_instance->get(tid));
    }
}


void test_class_launch(){

    const size_t N = 16;

    // Construct on the host
    TestClass * h_instance = new TestClass(N);

    // Construct.
    printf("construct...\n");
    h_instance->allocate();

    printf("h_instance %p \n", h_instance);

    // Launch a kernel with the instance as the parameter

    printf("kernel...\n");
    test_kernel<<<N, 1>>>(N, h_instance);
    CUDACHECK(hipDeviceSynchronize());
    printf("synced...\n");


    // Free
    printf("free...\n");
    h_instance->free();
    delete h_instance;

}
int main(int argc, char * argv[]){
    printf("main\n");

    launch2dexample();    

    return 1;
}
