#include <stdio.h>
#include <hip/hip_runtime.h>

class TestClass {

public:
    int * data;
    size_t len;



    TestClass(size_t len) {
        printf("Constructor\n");
        this->data = nullptr;
        this->len = len;
    }

    ~TestClass(){
        printf("~Destructor\n");
    }

    __host__ void allocate(){
        hipMalloc((void**) &this->data, this->len * sizeof(int));
        hipMemset(this->data, 0, this->len * sizeof(int));
    }

    __host__ void free(){
        hipFree(this->data);
        this->data = nullptr;
    }

    __device__ int get(size_t index){
        return this->data[index];
    }
    __device__ void set(size_t index, int value){
        this->data[index] = value;
    }

};


__global__ void test_kernel(unsigned int threads, TestClass * d_instance){
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < threads){
        // printf("Thread %u\n", tid);
        printf("Thread %u: d_isntance %p, element %d\n", tid, d_instance, d_instance->get(tid));
    }
}


int main(int argc, char * argv[]){
    printf("main\n");

    const size_t N = 16;

    // Construct on the host
    TestClass * h_instance = new TestClass(N);

    // Construct.
    printf("construct...\n");
    h_instance->allocate();

    printf("h_instance %p \n", h_instance);

    // Launch a kernel with the instance as the parameter

    printf("kernel...\n");
    test_kernel<<<N, 1>>>(N, h_instance);
    hipDeviceSynchronize();
    printf("synced...\n");


    // Free
    printf("free...\n");
    h_instance->free();
    delete h_instance;

    return 1;
}
