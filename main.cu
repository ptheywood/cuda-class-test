#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDACHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

class TestClass {

public:
    int * data;
    size_t len;



    TestClass(size_t len) {
        printf("Constructor\n");
        this->data = nullptr;
        this->len = len;
    }

    ~TestClass(){
        printf("~Destructor\n");
    }

    __host__ void allocate(){
        CUDACHECK(hipMalloc((void**) &this->data, this->len * sizeof(int)));
        CUDACHECK(hipMemset(this->data, 0, this->len * sizeof(int)));
    }

    __host__ void free(){
        CUDACHECK(hipFree(this->data));
        this->data = nullptr;
    }

    __device__ int get(size_t index){
        return this->data[index];
    }
    __device__ void set(size_t index, int value){
        this->data[index] = value;
    }

};




__global__ void test_kernel(unsigned int threads, TestClass * d_instance){
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < threads){
        // printf("Thread %u\n", tid);
        printf("Thread %u: d_isntance %p, element %d\n", tid, d_instance, d_instance->get(tid));
    }
}


void test_class_launch(){

    const size_t N = 16;

    // Construct on the host
    TestClass * h_instance = new TestClass(N);

    // Construct.
    printf("construct...\n");
    h_instance->allocate();

    printf("h_instance %p \n", h_instance);

    // Launch a kernel with the instance as the parameter

    printf("kernel...\n");
    test_kernel<<<N, 1>>>(N, h_instance);
    CUDACHECK(hipDeviceSynchronize());
    printf("synced...\n");


    // Free
    printf("free...\n");
    h_instance->free();
    delete h_instance;

}
int main(int argc, char * argv[]){
    printf("main\n");

    test_class_launch();    

    return 1;
}
